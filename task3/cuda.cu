#include "hip/hip_runtime.h"

#include <vector>
#include <iostream>



#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    exit(1);
  }
}

__global__ 
void cuda_test(float* left, float* right, float* result)
{
  int idx = threadIdx.x;
  
  result[idx] = left[idx] + right[idx];
}

void test() {
    int size = 128;
    std::vector<float> vec1 (size, 1.0);
    std::vector<float> vec2 (size, 3.0);
    std::vector<float> vec3 (size, 0.0);

    //Указатели на память видеокарте
    float* devVec1;
    float* devVec2;
    float* devVec3;

    checkCudaErrors(hipMalloc((void**)&devVec1, sizeof(float) * size));
    checkCudaErrors(hipMalloc((void**)&devVec2, sizeof(float) * size));
    checkCudaErrors(hipMalloc((void**)&devVec3, sizeof(float) * size));

    hipMemcpy(devVec1, vec1.data(), sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(devVec2, vec2.data(), sizeof(float) * size, hipMemcpyHostToDevice);

    dim3 gridSize = dim3(1, 1, 1);
    dim3 blockSize = dim3(size, 1, 1);

    //Выполняем вызов функции ядра
    cuda_test<<<gridSize, blockSize>>>(devVec1, devVec2, devVec3);

    hipEvent_t syncEvent;

    hipEventCreate(&syncEvent);    //Создаем event
    hipEventRecord(syncEvent, 0);  //Записываем event
    hipEventSynchronize(syncEvent);  //Синхронизируем event
  
    //Только теперь получаем результат расчета
    hipMemcpy(vec3.data(), devVec3, sizeof(float) * size, hipMemcpyDeviceToHost);

    std::cout << vec3[10] << std::endl;
}