#include "hip/hip_runtime.h"
#include "parallel.hpp"

#include <vector>
#include <iostream>
#include <math.h>

#include <hip/hip_runtime.h>

#include <mpi.h>

#define SAFE_CALL( CallInstruction ) { \
    hipError_t cuerr = CallInstruction; \
    if(cuerr != hipSuccess) { \
        printf("CUDA error: %s at call \"" #CallInstruction "\"\n", hipGetErrorString(cuerr)); \
            throw "error in CUDA API function, aborting..."; \
    } \
}

#define SAFE_KERNEL_CALL( KernelCallInstruction ) { \
    KernelCallInstruction; \
    hipError_t cuerr = hipGetLastError(); \
    if(cuerr != hipSuccess) { \
        printf("CUDA error in kernel launch: %s at kernel \"" #KernelCallInstruction "\"\n", hipGetErrorString(cuerr)); \
            throw "error in CUDA kernel launch, aborting..."; \
    } \
    cuerr = hipDeviceSynchronize(); \
    if(cuerr != hipSuccess) { \
        printf("CUDA error in kernel execution: %s at kernel \"" #KernelCallInstruction "\"\n", hipGetErrorString(cuerr)); \
            throw "error in CUDA kernel execution, aborting..."; \
    } \
}

double* d_block;

__constant__ double a_t;
__constant__ double L[3];
__constant__ double bshift[3];
__constant__ int bsize[3];
__constant__ int bmin[3];

__device__
double u(double x, double y, double z, double t) {
    return sin((M_PI / L[0]) * x) * 
           sin((M_PI / L[1]) * y) * 
           sin((M_PI / L[2]) * z) *
           cos(a_t * t);;
}

__global__
void u0(double* block)
{
    int i = blockIdx.z + 1;
    int j = blockIdx.y + 1;
    int k = threadIdx.x + 1;
    int ind = i * (bsize[1] * bsize[2]) + j * bsize[2] + k;
    block[ind] = 
        u((i + bmin[0]) * bshift[0], (j + bmin[1]) * bshift[1], (k + bmin[2]) * bshift[2], 0);
}

void init_u0(Block &b, Function3D &u) {
    double h_a_t = u.a_t();
    std::vector<double> h_L = u.getL();
    std::vector<double> h_bshift = b.getShift();
    std::vector<int> h_bsize = b.getSize();
    std::vector<int> h_bmin = b.getMin();

    int h_size = h_bsize[0] * h_bsize[1] * h_bsize[2];

    SAFE_CALL(hipMalloc((void**)&d_block, sizeof(double) * h_size));

    hipMemcpyToSymbol(HIP_SYMBOL(a_t), &h_a_t, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(L), h_L.data(), sizeof(double) * 3);
    hipMemcpyToSymbol(HIP_SYMBOL(bshift), h_bshift.data(), sizeof(double) * 3);
    hipMemcpyToSymbol(HIP_SYMBOL(bsize), h_bsize.data(), sizeof(int) * 3);
    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(bmin), h_bmin.data(), sizeof(int) * 3));

    dim3 grid = dim3(1, (h_bsize[1] - 2), (h_bsize[0] - 2));
    dim3 block = dim3(h_bsize[2] - 2, 1, 1);

    u0<<<grid, block>>>(d_block);

    SAFE_CALL(hipMemcpy(b.getData().data(), d_block, sizeof(double) * h_size, hipMemcpyDeviceToHost));
}

__device__
double getElem(double* block, int i, int j, int k) {
    return block[i * (bsize[1] * bsize[2]) + j * bsize[2] + k];
}

__device__
double lap_h(double* block, int i, int j, int k) {
    double central = getElem(block, i, j, k);
    return (getElem(block, i - 1, j, k) - 2 * central + getElem(block, i + 1, j, k)) / pow(bshift[0], 2) + 
           (getElem(block, i, j - 1, k) - 2 * central + getElem(block, i, j + 1, k)) / pow(bshift[1], 2) +
           (getElem(block, i, j, k - 1) - 2 * central + getElem(block, i, j, k + 1)) / pow(bshift[2], 2); 
}

__global__
void u1(double* block, double* u0, double tau) {
    int i = blockIdx.z + 1;
    int j = blockIdx.y + 1;
    int k = threadIdx.x + 1;
    int ind = i * (bsize[1] * bsize[2]) + j * bsize[2] + k;
    block[ind] = u0[ind] + (pow(tau, 2) / 2) * lap_h(u0, i, j, k);
}

void init_u1(Block &b, const Block &u0, double tau, Function3D &u) {
    std::vector<double> h_bshift = b.getShift();
    std::vector<int> h_bsize = b.getSize();

    int h_size = h_bsize[0] * h_bsize[1] * h_bsize[2];

    double* d_u0;

    SAFE_CALL(hipMalloc((void**)&d_u0, sizeof(double) * h_size));

    SAFE_CALL(hipMemcpy(d_u0, u0.getValData().data(), sizeof(double) * h_size, hipMemcpyHostToDevice));

    hipMemcpyToSymbol(HIP_SYMBOL(bshift), h_bshift.data(), sizeof(double) * 3);
    hipMemcpyToSymbol(HIP_SYMBOL(bsize), h_bsize.data(), sizeof(int) * 3);

    dim3 grid = dim3(1, (h_bsize[1] - 2), (h_bsize[0] - 2));
    dim3 block = dim3(h_bsize[2] - 2, 1, 1);

    u1<<<grid, block>>>(d_block, d_u0, tau);

    SAFE_CALL(hipMemcpy(b.getData().data(), d_block, sizeof(double) * h_size, hipMemcpyDeviceToHost));
    hipFree(d_u0);
}

__global__
void global_step(double* block, double* u1, double* u0, double tau) {
    int i = blockIdx.z + 1;
    int j = blockIdx.y + 1;
    int k = threadIdx.x + 1;
    int ind = i * (bsize[1] * bsize[2]) + j * bsize[2] + k;
    block[ind] = 2 * u1[ind] - u0[ind] + pow(tau, 2) * lap_h(u1, i, j, k);
}

void step(Block &b, const Block& u1, const Block& u0, double tau, Function3D &u) {
    std::vector<double> h_bshift = b.getShift();
    std::vector<int> h_bsize = b.getSize();

    int h_size = h_bsize[0] * h_bsize[1] * h_bsize[2];

    double* d_u1;
    double* d_u0;

    SAFE_CALL(hipMalloc((void**)&d_u1, sizeof(double) * h_size));
    SAFE_CALL(hipMalloc((void**)&d_u0, sizeof(double) * h_size));

    SAFE_CALL(hipMemcpy(d_u1, u1.getValData().data(), sizeof(double) * h_size, hipMemcpyHostToDevice));
    SAFE_CALL(hipMemcpy(d_u0, u0.getValData().data(), sizeof(double) * h_size, hipMemcpyHostToDevice));

    hipMemcpyToSymbol(HIP_SYMBOL(bshift), h_bshift.data(), sizeof(double) * 3);
    hipMemcpyToSymbol(HIP_SYMBOL(bsize), h_bsize.data(), sizeof(int) * 3);

    dim3 grid = dim3(1, (h_bsize[1] - 2), (h_bsize[0] - 2));
    dim3 block = dim3(h_bsize[2] - 2, 1, 1);

    global_step<<<grid, block>>>(d_block, d_u1, d_u0, tau);

    SAFE_CALL(hipMemcpy(b.getData().data(), d_block, sizeof(double) * h_size, hipMemcpyDeviceToHost));
    hipFree(d_u1);
    hipFree(d_u0);
}

__global__
void calcErrorK(double* d_error, double* block, double t) {
    extern __shared__ double sdata[];

    int i = blockIdx.z + 1;
    int j = blockIdx.y + 1;
    int k = threadIdx.x + 1;
    int tid = threadIdx.x;
    int ind = i * (bsize[1] * bsize[2]) + j * bsize[2] + k;

    sdata[tid] = fabs(block[ind] - u((i + bmin[0]) * bshift[0], (j + bmin[1]) * bshift[1], (k + bmin[2]) * bshift[2], t));
    __syncthreads();

    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (tid < s) {
            sdata[tid] = fmax(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();
    }

    if (tid == 0) d_error[blockIdx.z * gridDim.y + blockIdx.y] = sdata[0];
}

__global__
void calcErrorJ(double* d_errorJ, double* d_errorK) {
    extern __shared__ double sdata[];

    int tid = threadIdx.x;
    int ind = blockDim.x * blockIdx.y + threadIdx.x;

    sdata[tid] = d_errorK[ind];
    __syncthreads();

    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (tid < s) {
            sdata[tid] = fmax(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();
    }

    if (tid == 0) d_errorJ[blockIdx.y] = sdata[0];
}

__global__
void calcErrorI(double* d_errorJ) {
    extern __shared__ double sdata[];

    int tid = threadIdx.x;

    sdata[tid] = d_errorJ[tid];
    __syncthreads();

    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (tid < s) {
            sdata[tid] = fmax(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();
    }

    if (tid == 0) d_errorJ[0] = sdata[0];
}

double getError(const Block &b, Function3D &u, double t) {
    double h_a_t = u.a_t();
    std::vector<double> h_L = u.getL();
    std::vector<double> h_bshift = b.getShift();
    std::vector<int> h_bsize = b.getSize();
    std::vector<int> h_bmin = b.getMin();

    int h_size = h_bsize[0] * h_bsize[1] * h_bsize[2];

    double h_error;

    dim3 grid;
    dim3 block;

    double* d_errorK;
    double* d_errorJ;

    SAFE_CALL(hipMalloc((void**)&d_errorK, sizeof(double) * (h_bsize[0] - 2) * (h_bsize[1] - 2)));
    SAFE_CALL(hipMalloc((void**)&d_errorJ, sizeof(double) * (h_bsize[0] - 2)));

    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(a_t), &h_a_t, sizeof(double)));
    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(L), h_L.data(), sizeof(double) * 3));
    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(bshift), h_bshift.data(), sizeof(double) * 3));
    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(bsize), h_bsize.data(), sizeof(int) * 3));
    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(bmin), h_bmin.data(), sizeof(int) * 3));

    grid = dim3(1, (h_bsize[1] - 2), (h_bsize[0] - 2));
    block = dim3(h_bsize[2] - 2, 1, 1);

    calcErrorK<<<grid, block, block.x * sizeof(double)>>>(d_errorK, d_block, t);

    grid = dim3(1, (h_bsize[0] - 2), 1);
    block = dim3(h_bsize[1] - 2, 1, 1);

    calcErrorJ<<<grid, block, block.x * sizeof(double)>>>(d_errorJ, d_errorK);

    grid = dim3(1, 1, 1);
    block = dim3(h_bsize[0] - 2, 1, 1);

    calcErrorI<<<grid, block, block.x * sizeof(double)>>>(d_errorJ);

    SAFE_CALL(hipMemcpy(&h_error, d_errorJ, sizeof(double), hipMemcpyDeviceToHost));
    hipFree(d_errorK);
    hipFree(d_errorJ);
    return h_error;
}